#include "hip/hip_runtime.h"
#include "gpu_thread_pool.hpp"

typedef double FloatingPoint_t;

struct common_data_t {
	double * A;
	double * B;
	double * C;
	double alpha;
	double beta;
	int m;
	int n;
	int k;
	int lda;
	int ldb;
	int ldc;
	int offsetA;
	int offsetB;
};

struct task_t {
	int by, bx;
};

static inline __device__
double tex_fetch(texture<int2> tex_ref, int coord) {
	int2 v = tex1Dfetch(tex_ref, coord);
	return __hiloint2double(v.y, v.x);
}

texture<int2, 0x01, hipReadModeElementType> tex_ref_A;
texture<int2, 0x01, hipReadModeElementType> tex_ref_B;

template < >
__device__
void do_task < task_t, common_data_t, 0 > ( task_t const & task, common_data_t const & common_data ) {
    int M = common_data.m;
    int N = common_data.n;
    int K = common_data.k;
//  double * A = common_data.A;
    int LDA = common_data.lda;
//  double * B = common_data.B;
    int LDB = common_data.ldb;
    double * C = common_data.C;
    int LDC = common_data.ldc;
    double alpha = common_data.alpha;
    double beta = common_data.beta;
    int offsetA = common_data.offsetA;
    int offsetB = common_data.offsetB;

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    int idt = 16 * idy + idx;

    int idxA = idt % 16;
    int idyA = idt / 16;

    int idxB = idt % 16;
    int idyB = idt / 16;

//  int blx = blockIdx.x;
    int blx = task.bx;
//  int bly = blockIdx.y;
    int bly = task.by;

    __shared__ double sA[16][64 +1];
    __shared__ double sB[64][16 +1];


    double rC[( 64 / 16 )][( 64 / 16 )];
    double rA[( 64 / 16 )];
    double rB[( 64 / 16 )];





        double ra[16/16][64/16];




        double rb[64/16][16/16];






            int coord_A = offsetA + blx*64 + idyA*LDA+idxA;




            int coord_B = offsetB + bly*64*LDB + idyB*LDB+idxB;
# 165 "gemm_stencil.cu"
    int m, n, k, kk;


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
      for (m = 0; m < ( 64 / 16 ); m++)
        rC[n][m] = (0.0);
# 182 "gemm_stencil.cu"
#pragma unroll
        for (n = 0; n < 16; n += 16)
#pragma unroll
          for (m = 0; m < 64; m += 16)
            sA[n+idyA][m+idxA] = tex_fetch(tex_ref_A, coord_A + n*LDA+m);
# 197 "gemm_stencil.cu"
#pragma unroll
        for (n = 0; n < 64; n += 16)
#pragma unroll
          for (m = 0; m < 16; m += 16)
            sB[n+idyB][m+idxB] = tex_fetch(tex_ref_B, coord_B + n*LDB+m);


    __syncthreads();

    for (kk = 0; kk < K-16; kk += 16)
    {




                coord_A += 16*LDA;




                coord_B += 16;
# 240 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                ra[n][m] = tex_fetch(tex_ref_A, coord_A + n*16*LDA+m*16);
# 255 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                rb[n][m] = tex_fetch(tex_ref_B, coord_B + n*16*LDB+m*16);



#pragma unroll
        for (k = 0; k < 16; k++)
        {

#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
                rA[m] = sA[k][m*16 +idx];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
                rB[n] = sB[n*16 +idy][k];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
                for (m = 0; m < ( 64 / 16 ); m++)
# 291 "gemm_stencil.cu"
                        rC[n][m] += (rA[m]*rB[n]);


        }

        __syncthreads();
# 306 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                sA[n*16 +idyA][m*16 +idxA] = ra[n][m];
# 321 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                sB[n*16 +idyB][m*16 +idxB] = rb[n][m];


        __syncthreads();
    }


#pragma unroll
    for (k = 0; k < 16; k++)
    {

#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++)
            rA[m] = sA[k][m*16 +idx];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
            rB[n] = sB[n*16 +idy][k];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
# 360 "gemm_stencil.cu"
                    rC[n][m] += (rA[m]*rB[n]);


    }


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++) {
        int coord_dCn = bly*64 + n*16 +idy;
#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++) {
            int coord_dCm = blx*64 + m*16 +idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + coord_dCm;

                double &regC = rC[n][m];
                double &memC = C[offsC];

                memC = ((alpha*regC)+(beta*memC));
            }
        }
    }

}

template < >
__device__
void do_task < task_t, common_data_t, 1 > ( task_t const & task, common_data_t const & common_data ) {
    int M = common_data.m;
    int N = common_data.n;
    int K = common_data.k;
//  double * A = common_data.A;
    int LDA = common_data.lda;
//  double * B = common_data.B;
    int LDB = common_data.ldb;
    double * C = common_data.C;
    int LDC = common_data.ldc;
    double alpha = common_data.alpha;
    double beta = common_data.beta;
    int offsetA = common_data.offsetA;
    int offsetB = common_data.offsetB;

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    int idt = 16 * idy + idx;

    int idxA = idt % 16;
    int idyA = idt / 16;

    int idxB = idt % 16;
    int idyB = idt / 16;

//  int blx = blockIdx.x;
    int blx = task.bx;
//  int bly = blockIdx.y;
    int bly = task.by;

    __shared__ double sA[16][64 +1];
    __shared__ double sB[64][16 +1];


    FloatingPoint_t rC[( 64 / 16 )][( 64 / 16 )];
    FloatingPoint_t rA[( 64 / 16 )];
    FloatingPoint_t rB[( 64 / 16 )];





        FloatingPoint_t ra[16/16][64/16];


        FloatingPoint_t rb[16/16][64/16];
# 145 "gemm_stencil.cu"
            int coord_A = offsetA + blx*64 + idyA*LDA+idxA;


            int coord_B = offsetB + bly*64 + idyB*LDB+idxB;
# 165 "gemm_stencil.cu"
    int m, n, k, kk;


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
      for (m = 0; m < ( 64 / 16 ); m++)
        rC[n][m] = (0.0);
# 182 "gemm_stencil.cu"
#pragma unroll
        for (n = 0; n < 16; n += 16)
#pragma unroll
          for (m = 0; m < 64; m += 16)
            sA[n+idyA][m+idxA] = tex_fetch(tex_ref_A, coord_A + n*LDA+m);




#pragma unroll
        for (n = 0; n < 16; n += 16)
#pragma unroll
          for (m = 0; m < 64; m += 16)
            sB[m+idxB][n+idyB] = tex_fetch(tex_ref_B, coord_B + n*LDB+m);
# 204 "gemm_stencil.cu"
    __syncthreads();

    for (kk = 0; kk < K-16; kk += 16)
    {




                coord_A += 16*LDA;


                coord_B += 16*LDB;
# 240 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                ra[n][m] = tex_fetch(tex_ref_A, coord_A + n*16*LDA+m*16);




#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                rb[n][m] = tex_fetch(tex_ref_B, coord_B + n*16*LDB+m*16);
# 263 "gemm_stencil.cu"
#pragma unroll
        for (k = 0; k < 16; k++)
        {

#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
                rA[m] = sA[k][m*16 +idx];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
                rB[n] = sB[n*16 +idy][k];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
                for (m = 0; m < ( 64 / 16 ); m++)
# 291 "gemm_stencil.cu"
                        rC[n][m] += (rA[m]*rB[n]);


        }

        __syncthreads();
# 306 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                sA[n*16 +idyA][m*16 +idxA] = ra[n][m];




#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                sB[m*16 +idxB][n*16 +idyB] = rb[n][m];
# 328 "gemm_stencil.cu"
        __syncthreads();
    }


#pragma unroll
    for (k = 0; k < 16; k++)
    {

#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++)
            rA[m] = sA[k][m*16 +idx];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
            rB[n] = sB[n*16 +idy][k];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
# 360 "gemm_stencil.cu"
                    rC[n][m] += (rA[m]*rB[n]);


    }


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++) {
        int coord_dCn = bly*64 + n*16 +idy;
#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++) {
            int coord_dCm = blx*64 + m*16 +idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + coord_dCm;

                FloatingPoint_t &regC = rC[n][m];
                FloatingPoint_t &memC = C[offsC];

                memC = ((alpha*regC)+(beta*memC));
            }
        }
    }

}

template < >
__device__
void do_task < task_t, common_data_t, 2 > ( task_t const & task, common_data_t const & common_data ) {
    int M = common_data.m;
    int N = common_data.n;
    int K = common_data.k;
//  double * A = common_data.A;
    int LDA = common_data.lda;
//  double * B = common_data.B;
    int LDB = common_data.ldb;
    double * C = common_data.C;
    int LDC = common_data.ldc;
    double alpha = common_data.alpha;
    double beta = common_data.beta;
    int offsetA = common_data.offsetA;
    int offsetB = common_data.offsetB;

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    int idt = 16 * idy + idx;

    int idxA = idt % 16;
    int idyA = idt / 16;

    int idxB = idt % 16;
    int idyB = idt / 16;

//  int blx = blockIdx.x;
    int blx = task.bx;
//  int bly = blockIdx.y;
    int bly = task.by;

    __shared__ double sA[16][64 +1];
    __shared__ double sB[64][16 +1];

    FloatingPoint_t rC[( 64 / 16 )][( 64 / 16 )];
    FloatingPoint_t rA[( 64 / 16 )];
    FloatingPoint_t rB[( 64 / 16 )];



        FloatingPoint_t ra[64/16][16/16];






        FloatingPoint_t rb[64/16][16/16];




            int coord_A = offsetA + blx*64*LDA + idyA*LDA+idxA;






            int coord_B = offsetB + bly*64*LDB + idyB*LDB+idxB;
# 165 "gemm_stencil.cu"
    int m, n, k, kk;


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
      for (m = 0; m < ( 64 / 16 ); m++)
        rC[n][m] = (0.0);



#pragma unroll
        for (n = 0; n < 64; n += 16)
#pragma unroll
          for (m = 0; m < 16; m += 16)
            sA[m+idxA][n+idyA] = tex_fetch(tex_ref_A, coord_A + n*LDA+m);
# 197 "gemm_stencil.cu"
#pragma unroll
        for (n = 0; n < 64; n += 16)
#pragma unroll
          for (m = 0; m < 16; m += 16)
            sB[n+idyB][m+idxB] = tex_fetch(tex_ref_B, coord_B + n*LDB+m);


    __syncthreads();

    for (kk = 0; kk < K-16; kk += 16)
    {


                coord_A += 16;






                coord_B += 16;
# 234 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                ra[n][m] = tex_fetch(tex_ref_A, coord_A + n*16*LDA+m*16);
# 255 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                rb[n][m] = tex_fetch(tex_ref_B, coord_B + n*16*LDB+m*16);



#pragma unroll
        for (k = 0; k < 16; k++)
        {

#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
                rA[m] = sA[k][m*16 +idx];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
                rB[n] = sB[n*16 +idy][k];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
                for (m = 0; m < ( 64 / 16 ); m++)
# 291 "gemm_stencil.cu"
                        rC[n][m] += (rA[m]*rB[n]);


        }

        __syncthreads();



#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                sA[m*16 +idxA][n*16 +idyA] = ra[n][m];
# 321 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                sB[n*16 +idyB][m*16 +idxB] = rb[n][m];


        __syncthreads();
    }


#pragma unroll
    for (k = 0; k < 16; k++)
    {

#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++)
            rA[m] = sA[k][m*16 +idx];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
            rB[n] = sB[n*16 +idy][k];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
# 360 "gemm_stencil.cu"
                    rC[n][m] += (rA[m]*rB[n]);


    }


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++) {
        int coord_dCn = bly*64 + n*16 +idy;
#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++) {
            int coord_dCm = blx*64 + m*16 +idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + coord_dCm;

                FloatingPoint_t &regC = rC[n][m];
                FloatingPoint_t &memC = C[offsC];

                memC = ((alpha*regC)+(beta*memC));
            }
        }
    }

}

template < >
__device__
void do_task < task_t, common_data_t, 3 > ( task_t const & task, common_data_t const & common_data ) {
    int M = common_data.m;
    int N = common_data.n;
    int K = common_data.k;
//  double * A = common_data.A;
    int LDA = common_data.lda;
//  double * B = common_data.B;
    int LDB = common_data.ldb;
    double * C = common_data.C;
    int LDC = common_data.ldc;
    double alpha = common_data.alpha;
    double beta = common_data.beta;
    int offsetA = common_data.offsetA;
    int offsetB = common_data.offsetB;

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    int idt = 16 * idy + idx;

    int idxA = idt % 16;
    int idyA = idt / 16;

    int idxB = idt % 16;
    int idyB = idt / 16;

//  int blx = blockIdx.x;
    int blx = task.bx;
//  int bly = blockIdx.y;
    int bly = task.by;

    __shared__ double sA[16][64 +1];
    __shared__ double sB[64][16 +1];


    FloatingPoint_t rC[( 64 / 16 )][( 64 / 16 )];
    FloatingPoint_t rA[( 64 / 16 )];
    FloatingPoint_t rB[( 64 / 16 )];



        FloatingPoint_t ra[64/16][16/16];




        FloatingPoint_t rb[16/16][64/16];






            int coord_A = offsetA + blx*64*LDA + idyA*LDA+idxA;




            int coord_B = offsetB + bly*64 + idyB*LDB+idxB;
# 165 "gemm_stencil.cu"
    int m, n, k, kk;


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
      for (m = 0; m < ( 64 / 16 ); m++)
        rC[n][m] = (0.0);



#pragma unroll
        for (n = 0; n < 64; n += 16)
#pragma unroll
          for (m = 0; m < 16; m += 16)
            sA[m+idxA][n+idyA] = tex_fetch(tex_ref_A, coord_A + n*LDA+m);
# 191 "gemm_stencil.cu"
#pragma unroll
        for (n = 0; n < 16; n += 16)
#pragma unroll
          for (m = 0; m < 64; m += 16)
            sB[m+idxB][n+idyB] = tex_fetch(tex_ref_B, coord_B + n*LDB+m);
# 204 "gemm_stencil.cu"
    __syncthreads();

    for (kk = 0; kk < K-16; kk += 16)
    {


                coord_A += 16;




                coord_B += 16*LDB;
# 234 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                ra[n][m] = tex_fetch(tex_ref_A, coord_A + n*16*LDA+m*16);
# 249 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                rb[n][m] = tex_fetch(tex_ref_B, coord_B + n*16*LDB+m*16);
# 263 "gemm_stencil.cu"
#pragma unroll
        for (k = 0; k < 16; k++)
        {

#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
                rA[m] = sA[k][m*16 +idx];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
                rB[n] = sB[n*16 +idy][k];


#pragma unroll
            for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
                for (m = 0; m < ( 64 / 16 ); m++)
# 291 "gemm_stencil.cu"
                        rC[n][m] += (rA[m]*rB[n]);


        }

        __syncthreads();



#pragma unroll
            for (n = 0; n < 64/16; n++)
#pragma unroll
              for (m = 0; m < 16/16; m++)
                sA[m*16 +idxA][n*16 +idyA] = ra[n][m];
# 315 "gemm_stencil.cu"
#pragma unroll
            for (n = 0; n < 16/16; n++)
#pragma unroll
              for (m = 0; m < 64/16; m++)
                sB[m*16 +idxB][n*16 +idyB] = rb[n][m];
# 328 "gemm_stencil.cu"
        __syncthreads();
    }


#pragma unroll
    for (k = 0; k < 16; k++)
    {

#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++)
            rA[m] = sA[k][m*16 +idx];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
            rB[n] = sB[n*16 +idy][k];


#pragma unroll
        for (n = 0; n < ( 64 / 16 ); n++)
#pragma unroll
            for (m = 0; m < ( 64 / 16 ); m++)
# 360 "gemm_stencil.cu"
                    rC[n][m] += (rA[m]*rB[n]);


    }


#pragma unroll
    for (n = 0; n < ( 64 / 16 ); n++) {
        int coord_dCn = bly*64 + n*16 +idy;
#pragma unroll
        for (m = 0; m < ( 64 / 16 ); m++) {
            int coord_dCm = blx*64 + m*16 +idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + coord_dCm;

                FloatingPoint_t &regC = rC[n][m];
                FloatingPoint_t &memC = C[offsC];

                memC = ((alpha*regC)+(beta*memC));
            }
        }
    }

}
